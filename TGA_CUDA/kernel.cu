#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc_c.h"
#include <time.h>
#include <iostream>
#include <stdio.h>
#include <hip/device_functions.h>
#include <iostream>
#include <string.h>
#ifndef __HIPCC__  
#define __HIPCC__
#endif

const unsigned int bSize = 128;

using namespace cv;

hipEvent_t cStart, cEnd;
#define CUDA_TIME_START() hipEventCreate(&cStart); hipEventCreate(&cEnd); hipEventRecord(cStart);
#define CUDA_TIME_GET(_ms) hipEventRecord(cEnd); hipEventSynchronize(cEnd); hipEventElapsedTime(&_ms,cStart, cEnd);
clock_t tBegin;
#define TIME_START() { tBegin = clock();}
#define TIME_GET() (double)(clock() - tBegin)/(CLOCKS_PER_SEC/1000);

#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
#define pb(bte){printf("%d\n",(bte));}
#define CUDA_ERROR_CHECK
inline void __cudaCheckError(const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}

	// More careful checking. However, this will affect performance.
	// Comment away if needed.
	err = hipDeviceSynchronize();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}
__global__ void sobel(unsigned char* imgray, unsigned char* out, int SIZE)
{

	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	unsigned char pixel00 = imgray[(x - 1) * SIZE + y - 1];
	unsigned char pixel01 = imgray[(x - 1) * SIZE + y];
	unsigned char pixel02 = imgray[(x - 1) * SIZE + y + 1];
	unsigned char pixel10 = imgray[(x)*     SIZE + y - 1];
	unsigned char pixel12 = imgray[(x)*     SIZE + y + 1];
	unsigned char pixel20 = imgray[(x + 1) * SIZE + y - 1];
	unsigned char pixel21 = imgray[(x + 1) * SIZE + y];
	unsigned char pixel22 = imgray[(x + 1) * SIZE + y + 1];
	int vert = (pixel00 + 2 * pixel01 + pixel02) - (pixel20 + 2 * pixel21 + pixel22);
	int hori = (pixel00 + 2 * pixel10 + pixel20) - (pixel02 + 2 * pixel12 + pixel22);
	int tot = vert + hori;
	tot = (tot>60) ? 255 : 0;
	out[x * SIZE + y] = tot;

}

/*La shared memory es a nivel de bloque*/
/*Kernel  = Conjunto de bloques
Block =  Conjunto de threads


16*16 grid	256 grids
32*32 bloques	1024 bloques
(en total� 512*512�)
*/
__global__ void sobelBlocks(unsigned char* imgray, unsigned char* out, int SIZE)
{

	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int sizeShared = bSize + 2;
	//__shared__ unsigned char sA[(bSize + 2)*(bSize + 2)];//creamos array que contiene toda la imagen, todos los threads participan al crearlo
	__shared__ unsigned char sA[(bSize + 2)][(bSize + 2)];
	sA[(tx + 1)][(ty + 1)] = imgray[x*SIZE + y];
	if (ty == 0) // primera columna
	{
		//calcular todos los pixels x-1
		sA[tx + 1][0] = imgray[(x - 1)*SIZE + y];
		if (tx == 0) sA[0][0] = imgray[(x - 1)*SIZE + y - 1];//cargar arriba izquierda
	}
	if (ty == blockDim.y - 1) //ultima columna
	{
		sA[tx + 1][blockDim.y + 1] = imgray[(x + 1)*SIZE + y];
		if (tx == blockDim.x - 1) sA[blockDim.x + 1][blockDim.y + 1] = imgray[(x + 1)*SIZE + y + 1];//cargar abajo derecha
	}
	if (tx == 0)// primera fila
	{
		sA[0][ty + 1] = imgray[(x)*SIZE + y - 1];
		if (ty == blockDim.y - 1) sA[0][blockDim.y + 1] = imgray[(x + 1)*SIZE + y - 1];//cargar arriba derecha
	}
	if (tx == blockDim.x - 1) //ultima fila
	{
		sA[blockDim.x + 1][ty + 1] = imgray[(x)*SIZE + y + 1];
		if (ty == 0) sA[blockDim.y + 1][0] = imgray[(x - 1)*SIZE + y + 1];//cargar abajo izquierda
	}
	__syncthreads();//ahora esperamos para que todos tengan una versi�n de la matriz en shared

	int tot;

	//boundary check? evitar que est� fuera del bloque?

	int ntx = tx + 1;
	int nty = ty + 1;

	unsigned char pixel00 = sA[(ntx - 1)][nty - 1];
	unsigned char pixel01 = sA[(ntx - 1)][nty];
	unsigned char pixel02 = sA[(ntx - 1)][nty + 1];
	unsigned char pixel10 = sA[(ntx)][nty - 1];
	unsigned char pixel12 = sA[(ntx)][nty + 1];
	unsigned char pixel20 = sA[(ntx + 1)][nty - 1];
	unsigned char pixel21 = sA[(ntx + 1)][+nty];
	unsigned char pixel22 = sA[(ntx + 1)][nty + 1];

	int vert = (pixel00 + 2 * pixel01 + pixel02) - (pixel20 + 2 * pixel21 + pixel22);
	int hori = (pixel00 + 2 * pixel10 + pixel20) - (pixel02 + 2 * pixel12 + pixel22);
	tot = vert + hori;
	tot = (tot > 60) ? 255 : 0;


	__syncthreads();
	out[x * SIZE + y] = tot;

}

__global__ void sobelBlocks_4(unsigned char* imgray, unsigned char* out, int SIZE)
{

	int x = blockDim.x*blockIdx.x * 4 + threadIdx.x * 4;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int tx = threadIdx.x * 4;
	int ty = threadIdx.y;
	
	__shared__ unsigned char sA[(bSize + 2)][(bSize + 2)];

	sA[(ty + 1)][(tx + 1)] = imgray[y*SIZE + x];
	sA[(ty + 1)][(tx + 2)] = imgray[y*SIZE + x + 1];
	sA[(ty + 1)][(tx + 3)] = imgray[y*SIZE + x + 2];
	sA[(ty + 1)][(tx + 4)] = imgray[y*SIZE + x + 3];
	//cada thread carga 4
	if (ty == 0) // primera fila
	{
		sA[0][tx + 1] = imgray[(y - 1)*SIZE + x];
		sA[0][tx + 2] = imgray[(y - 1)*SIZE + x + 1];
		sA[0][tx + 3] = imgray[(y - 1)*SIZE + x + 2];
		sA[0][tx + 4] = imgray[(y - 1)*SIZE + x + 3];
	}
	if (ty == blockDim.y - 1) //ultima fila
	{
		sA[blockDim.y + 1][tx + 1] = imgray[(y + 1)*SIZE + x];
		sA[blockDim.y + 1][tx + 2] = imgray[(y + 1)*SIZE + x + 1];
		sA[blockDim.y + 1][tx + 3] = imgray[(y + 1)*SIZE + x + 2];
		sA[blockDim.y + 1][tx + 4] = imgray[(y + 1)*SIZE + x + 3];

	}
	if (tx == 0)// primera columna
	{
		sA[ty + 1][0] = imgray[(y - 1)*SIZE + x];

	}
	if (threadIdx.x == blockDim.x - 1) //ultima columna
	{
		sA[ty + 1][blockDim.x * 4 + 1] = imgray[(y + 1)*SIZE + x + 4];
	}
	sA[0][0] = sA[0][1];
	sA[blockDim.y + 1][blockDim.y + 1] = sA[blockDim.y][blockDim.y];
	sA[0][blockDim.y + 1] = sA[0][blockDim.y];
	sA[blockDim.y + 1][0] = sA[blockDim.y][0];
	__syncthreads();

	int ntx = tx + 1;
	int nty = ty + 1;
	uchar4 rest;
	int tot;

	unsigned char pixel00 = sA[(nty - 1)][ntx - 1];
	unsigned char pixel01 = sA[(nty - 1)][ntx];
	unsigned char pixel02 = sA[(nty - 1)][ntx + 1];
	unsigned char pixel10 = sA[nty][ntx - 1];
	unsigned char pixel11 = sA[nty][ntx];
	unsigned char pixel12 = sA[nty][ntx + 1];
	unsigned char pixel20 = sA[(nty + 1)][ntx - 1];
	unsigned char pixel21 = sA[(nty + 1)][ntx];
	unsigned char pixel22 = sA[(nty + 1)][ntx + 1];


	int vert = (pixel00 + 2 * pixel01 + pixel02) - (pixel20 + 2 * pixel21 + pixel22);
	int hori = (pixel00 + 2 * pixel10 + pixel20) - (pixel02 + 2 * pixel12 + pixel22);
	rest.x = ((vert + hori) > 60) ? 255 : 0;

	pixel00 = sA[(nty - 1)][ntx + 2];
	pixel10 = sA[nty][ntx + 2];
	pixel20 = sA[(nty + 1)][ntx + 2];

	vert = (pixel01 + 2 * pixel02 + pixel00) - (pixel21 + 2 * pixel22 + pixel20);
	hori = (pixel01 + 2 * pixel11 + pixel21) - (pixel00 + 2 * pixel10 + pixel20);
	rest.y = ((vert + hori) > 60) ? 255 : 0;

	pixel01 = sA[(nty - 1)][ntx + 3];
	pixel11 = sA[nty][ntx + 3];
	pixel21 = sA[(nty + 1)][ntx + 3];

	vert = (pixel02 + 2 * pixel00 + pixel01) - (pixel22 + 2 * pixel20 + pixel21);
	hori = (pixel02 + 2 * pixel12 + pixel22) - (pixel01 + 2 * pixel11 + pixel21);

	rest.z = ((vert + hori) > 60) ? 255 : 0;

	pixel02 = sA[(nty - 1)][ntx + 4];
	pixel12 = sA[nty][ntx + 4];
	pixel22 = sA[(nty + 1)][ntx + 4];
	vert = (pixel00 + 2 * pixel01 + pixel02) - (pixel20 + 2 * pixel21 + pixel22);
	hori = (pixel00 + 2 * pixel10 + pixel20) - (pixel02 + 2 * pixel12 + pixel22);

	rest.w = ((vert + hori) > 60) ? 255 : 0;
	__syncthreads();

	out[y * SIZE + x] = rest.x;
	out[y * SIZE + x + 1] = rest.y;
	out[y * SIZE + x + 2] = rest.z;
	out[y * SIZE + x + 3] = rest.w;
}



__global__ void asciiBlocks(unsigned char* imgray, unsigned char* out, int SIZEX, int bSizex, int bSizey) {

	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int sum = 0;
	/*
	for (int i = x*bSizex; i < x*bSizex+bSizex; i++)
	for (int j = y*bSizey; j < y*bSizey+bSizey; j++)
	sum = sum + imgray[j*SIZEX+i];

	*/
	__shared__ unsigned char sdata[8192];
	int tid = ty*SIZEX + tx;

	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = imgray[y*SIZEX + x];

	__syncthreads();

	// contiguous range pattern
	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
		if (threadIdx.x < offset) {
			// add a partial sum upstream to our own
			sdata[threadIdx.x] += sdata[threadIdx.x + offset];
		}
		// wait until all threads in the block have
		// updated their partial sums
		__syncthreads();
	}

	// thread 0 writes the final result
	if (threadIdx.x == 0) {
		//	per_block_results[blockIdx.x] = sdata[0];
	}



	//sdata[tid] = g_imgray[i];
	__syncthreads();





	__syncthreads();

	sum = sum / (bSizex*bSizey);
	unsigned char asciival = 'a';
	unsigned char value = sum;

	if (value >= 230)
	{
		asciival = '@';
	}
	else if (value >= 200)
	{
		asciival = '#';
	}
	else if (value >= 180)
	{
		asciival = '8';
	}
	else if (value >= 160)
	{
		asciival = '&';
	}
	else if (value >= 130)
	{
		asciival = 'o';
	}
	else if (value >= 100)
	{
		asciival = ':';
	}
	else if (value >= 70)
	{
		asciival = '*';
	}
	else if (value >= 50)
	{
		asciival = '.';
	}
	else
	{
		asciival = ' ';
	}

	int outSizex = SIZEX / bSizex;
	out[y*outSizex + x] = asciival;

}




__global__ void asciiMean(unsigned char* imgray, unsigned char* out, int SIZE, int bSizex, int bSizey) {

	int resx = blockIdx.x; //columna
	int resy = blockIdx.y; //fila
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int sum = 0;
	extern __shared__ int sdata[];//int para evitar overflow....

	sdata[tx] = imgray[x*SIZE + y];
	sdata[tx] = imgray[x*SIZE + y];
}



void CPUSobel(unsigned char* imgray, unsigned char* out, int SIZE)
{


	for (int x = 1; x<512; ++x)
		for (int y = 0; y < 511; ++y)
		{
			unsigned char pixel00 = imgray[(x - 1) * SIZE + y - 1];
			unsigned char pixel01 = imgray[(x - 1) * SIZE + y];
			unsigned char pixel02 = imgray[(x - 1) * SIZE + y + 1];
			unsigned char pixel10 = imgray[(x)* SIZE + y - 1];
			unsigned char pixel12 = imgray[(x)* SIZE + y + 1];
			unsigned char pixel20 = imgray[(x + 1) * SIZE + y - 1];
			unsigned char pixel21 = imgray[(x + 1) * SIZE + y];
			unsigned char pixel22 = imgray[(x + 1) * SIZE + y + 1];
			int vert = (pixel00 + 2 * pixel01 + pixel02) - (pixel20 + 2 * pixel21 + pixel22);
			int hori = (pixel00 + 2 * pixel10 + pixel20) - (pixel02 + 2 * pixel12 + pixel22);
			int tot = vert + hori;
			tot = (tot>60) ? 255 : 0;
			out[x * SIZE + y] = tot;
		}

}
unsigned char convertTable(unsigned char value)
{
	unsigned char asciival;

	if (value >= 230)
	{
		asciival = '@';
	}
	else if (value >= 200)
	{
		asciival = '#';
	}
	else if (value >= 180)
	{
		asciival = '8';
	}
	else if (value >= 160)
	{
		asciival = '&';
	}
	else if (value >= 130)
	{
		asciival = 'o';
	}
	else if (value >= 100)
	{
		asciival = ':';
	}
	else if (value >= 70)
	{
		asciival = '*';
	}
	else if (value >= 50)
	{
		asciival = '.';
	}
	else
	{
		asciival = ' ';
	}

	return asciival;
}
void CPUAscii(unsigned char* imgray, int SIZE, int cols, int rows)
{


	cols = 128;
	rows = 32;

	int  pixels_y = SIZE / cols;
	int pixels_x = SIZE / rows;
	//printf("pixelx=%d, pixely=%d SIZE=%d ", pixels_x, pixels_y, SIZE);
	//printf("Cols:%d Rows:%d", cols, rows);
	unsigned char* ascii = (unsigned char*)malloc(rows*cols+1);
	volatile int eol = 0;

		for (int x = 0; x < rows; x++)
		{
			for (int y = 0; y < cols; y++)
			{

				int sumt = 0;
				int dval = 1;
				for (int i = x*pixels_x; i < x*pixels_x + pixels_x; ++i)
				{
					for (int j = y*pixels_y; j < y*pixels_y + pixels_y; ++j)
					{
						++dval;
						sumt += imgray[i*SIZE + j];
					}
				}
					
				if (dval == 0) dval = 1;
				int media = sumt / dval;
				ascii[x*cols + y] = convertTable(media);
			}
		}
	
		ascii[rows*cols] = 0;
	printf((char*)ascii);
	printf("\n\n");


}
float serial()
{
	
	
	IplImage* image;
	image = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);
	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;

	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	TIME_START();
	CPUSobel(input, output, cvGetSize(image).height);
	return TIME_GET();
}
void ASCII()
{
	IplImage* image;
	image = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);
	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;

	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;

	CPUSobel(input, output, cvGetSize(image).height);
	CPUAscii((unsigned char*)h_image2->imageData, cvGetSize(image).height, 207, 61);

	cvShowImage("Image", h_image2);
	cvWaitKey();
}
void cudaASCII() {

	IplImage* image;
	image = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);
	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;

	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;

	CPUSobel(input, output, cvGetSize(image).height);


	//CONSOLE_SCREEN_BUFFER_INFO csbi;
	int a; std::cin >> a;

	int cols = 128;
	int rows = 32;

	std::cout << cols << " g " << rows;
	unsigned char *ascii = (unsigned char*)malloc(rows*cols);

	int SIZE = cvGetSize(image).height;

	int pixels_x = SIZE / rows; // character sizeX
	int pixels_y = SIZE / cols; //character sizeY
	int asciisize = rows*cols;
	unsigned char *d_input;
	unsigned char *d_output;

	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, asciisize);

	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//thread x block GRID
	int x, y;
	x = y = 2;
	while (x < rows)
		x *= 2;
	while (y < cols)
		y *= 2;

	//thread x block GRID
	dim3 dimBlock(pixels_x, pixels_y);
	dim3 dimGrid(rows, cols);

	//thread x pixel GRID
	//dim3 dimBlock(32, 32);

	float milis;
	CUDA_TIME_START();

	asciiBlocks << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height, pixels_x, pixels_y);
	CUDA_TIME_GET(milis);

	std::cout << "Milisegundos ejecuci�n CPU:" << milis << std::endl;
	CudaCheckError();


	//obtener los datos de la gr�fica

	hipMemcpy(ascii, d_output, asciisize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);

	//for (int i = 0; i < asciisize; i++)
	//std::cout << std::hex << (int)ascii[i];
	printf((char*)ascii);
	printf("\n");
	printf("hello");
	cvShowImage("Image", h_image2);
	cvWaitKey();
}


void mycuda()
{
	IplImage* image;

	image = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);

	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
	IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;
	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	unsigned char *d_input;
	unsigned char *d_output;


	//pinned memory host
	//hipHostMalloc((unsigned char**)&output, imgsize);
	//hipHostMalloc((unsigned char**)&input, imgsize);


	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, imgsize);


	//copiamos el input al device
	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
	dim3 dimBlock(32, 32);
	dim3 dimGrid(16, 16);


	float milis;
	CUDA_TIME_START();

	sobelBlocks << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);
	CUDA_TIME_GET(milis);

	//std::cout << "Milisegundos ejecuci�n CPU:" << milis << std::endl;
	CudaCheckError();


	//obtener los datos de la gr�fica
	hipMemcpy(output, d_output, imgsize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);
	//mostrar imagen

	cvShowImage("Image", h_image2);
	cvWaitKey();

}

/*	namedWindow("Image", WINDOW_NORMAL);
	cvShowImage("Image", h_image2);
	cvWaitKey();*/

float cuda128()
{
	IplImage* src;

	src = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);


	IplImage *image = cvCreateImage(cvSize(128, 128), src->depth, src->nChannels);


	cvResize(src, image);

	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
	IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;
	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	unsigned char *d_input;
	unsigned char *d_output;

	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, imgsize);


	//copiamos el input al device
	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
	dim3 dimBlock(16, 16);
	dim3 dimGrid(8, 8);

	float milis;
	CUDA_TIME_START();

	sobelBlocks << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);
	CUDA_TIME_GET(milis);

	//std::cout << "Milisegundos ejecuci�n CPU:" << milis << std::endl;
	CudaCheckError();


	//obtener los datos de la gr�fica
	hipMemcpy(output, d_output, imgsize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);
	//mostrar imagen
	return milis;
}
float cuda128_4()
{
	IplImage* src;

	src = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);


	IplImage *image = cvCreateImage(cvSize(128, 128), src->depth, src->nChannels);


	cvResize(src, image);

	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
	IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;
	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	unsigned char *d_input;
	unsigned char *d_output;

	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, imgsize);


	//copiamos el input al device
	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
	dim3 dimBlock(16, 4);
	dim3 dimGrid(8, 8);

	float milis;
	CUDA_TIME_START();

	sobelBlocks_4 << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);
	CUDA_TIME_GET(milis);

	//std::cout << "Milisegundos ejecuci�n CPU:" << milis << std::endl;
	CudaCheckError();


	//obtener los datos de la gr�fica
	hipMemcpy(output, d_output, imgsize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);
	//mostrar imagen
	return milis;

}
float cuda512()
{
	IplImage* src;

	src = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);


	IplImage *image = cvCreateImage(cvSize(512, 512), src->depth, src->nChannels);


	cvResize(src, image);

	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
	IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;
	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	unsigned char *d_input;
	unsigned char *d_output;

	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, imgsize);


	//copiamos el input al device
	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
	dim3 dimBlock(32, 32);
	dim3 dimGrid(16, 16);

	float milis;
	CUDA_TIME_START();

	sobelBlocks << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);
	CUDA_TIME_GET(milis);

	//std::cout << "Milisegundos ejecuci�n CPU:" << milis << std::endl;
	CudaCheckError();


	//obtener los datos de la gr�fica
	hipMemcpy(output, d_output, imgsize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);
	//mostrar imagen

	return milis;


}
float cuda512_4()
{
	IplImage* src;

	src = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);


	IplImage *image = cvCreateImage(cvSize(512, 512), src->depth, src->nChannels);


	cvResize(src, image);

	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
	IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;
	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	unsigned char *d_input;
	unsigned char *d_output;

	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, imgsize);


	//copiamos el input al device
	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
	dim3 dimBlock(8, 32);//x = 8
	dim3 dimGrid(16, 16);

	float milis;
	CUDA_TIME_START();

	sobelBlocks_4 << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);
	CUDA_TIME_GET(milis);

	//std::cout << "Milisegundos ejecuci�n CPU:" << milis << std::endl;
	CudaCheckError();


	//obtener los datos de la gr�fica
	hipMemcpy(output, d_output, imgsize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);
	//mostrar imagen

	//cvShowImage("Image", h_image2);
	//CPUAscii((unsigned char*)h_image2->imageData, cvGetSize(image).height, 207, 61);
	//cvWaitKey();
	return milis;

}
float cuda3072()
{
	IplImage* src;

	src = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);


	IplImage *image = cvCreateImage(cvSize(3072, 3072), src->depth, src->nChannels);


	cvResize(src, image);

	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
	IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;
	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	unsigned char *d_input;
	unsigned char *d_output;

	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, imgsize);


	//copiamos el input al device
	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
	dim3 dimBlock(32, 32);
	dim3 dimGrid(96, 96);

	float milis;
	CUDA_TIME_START();

	sobelBlocks << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);

	CUDA_TIME_GET(milis);

	//std::cout << "Milisegundos ejecuci�n CPU:" << milis << std::endl;
	CudaCheckError();


	//obtener los datos de la gr�fica
	hipMemcpy(output, d_output, imgsize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);
	//mostrar imagen

	return milis;

}
float cuda3072_4()
{
	IplImage* src;

	src = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);


	IplImage *image = cvCreateImage(cvSize(3072, 3072), src->depth, src->nChannels);


	cvResize(src, image);

	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
	IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;
	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	unsigned char *d_input;
	unsigned char *d_output;

	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, imgsize);


	//copiamos el input al device
	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
	dim3 dimBlock(8, 32);
	dim3 dimGrid(96, 96);

	float milis;
	CUDA_TIME_START();

	sobelBlocks_4 << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);

	CUDA_TIME_GET(milis);

	//std::cout << "Milisegundos ejecuci�n CPU:" << milis << std::endl;
	CudaCheckError();


	//obtener los datos de la gr�fica
	hipMemcpy(output, d_output, imgsize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);
	//mostrar imagen

	return milis;

}
float cuda4096()
{
	IplImage* src;

	src = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);


	IplImage *image = cvCreateImage(cvSize(4096, 4096), src->depth, src->nChannels);


	cvResize(src, image);

	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
	IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;
	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	unsigned char *d_input;
	unsigned char *d_output;

	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, imgsize);


	//copiamos el input al device
	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
	dim3 dimBlock(32, 32);
	dim3 dimGrid(128, 128);

	float milis;
	CUDA_TIME_START();

	sobelBlocks << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);
	CUDA_TIME_GET(milis);

	//std::cout << "Milisegundos ejecuci�n CPU:" << milis << std::endl;
	CudaCheckError();


	//obtener los datos de la gr�fica
	hipMemcpy(output, d_output, imgsize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);
	//mostrar imagen
	return milis;

}
float cuda4096_4()
{
	IplImage* src;

	src = cvLoadImage("cameraman.png", CV_LOAD_IMAGE_GRAYSCALE);


	IplImage *image = cvCreateImage(cvSize(4096, 4096), src->depth, src->nChannels);


	cvResize(src, image);

	IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
	IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

	int imgsize = cvGetSize(image).height* cvGetSize(image).width;
	unsigned char *output = (unsigned char*)h_image2->imageData;
	unsigned char *input = (unsigned char*)image->imageData;
	unsigned char *d_input;
	unsigned char *d_output;

	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, imgsize);
	hipMalloc((unsigned char**)&d_output, imgsize);


	//copiamos el input al device
	hipMemcpy(d_input, input, imgsize, hipMemcpyHostToDevice);

	//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
	dim3 dimBlock(8, 32);
	dim3 dimGrid(128, 128);

	float milis;
	CUDA_TIME_START();

	sobelBlocks_4 << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);
	CUDA_TIME_GET(milis);

	CudaCheckError();


	//obtener los datos de la gr�fica
	hipMemcpy(output, d_output, imgsize, hipMemcpyDeviceToHost);
	hipFree(d_output);
	hipFree(d_input);
	return milis;

}

void createVideoAscii()
{

	CvCapture* capture = cvCaptureFromAVI("E:\\Rewrite_t.mp4");
	if (capture == NULL) printf("Capture null");
	
	unsigned char *d_input;
	unsigned char *d_output;

	//reservamos espacio en la tg para nuestras imagenes
	hipMalloc((unsigned char**)&d_input, 512*512);
	hipMalloc((unsigned char**)&d_output, 512*512);
	while (1)
	{
		IplImage* frame = NULL;

		frame = cvQueryFrame(capture);
		if (frame == NULL)
		{
			int fafafa;
			printf("frame null\n");
			//scanf("%d", fafafa);
			break;
		}

		//transformamos la imagen
		IplImage* gray = cvCreateImage(cvGetSize(frame), IPL_DEPTH_8U, 1);
		cvCvtColor(frame, gray, CV_RGB2GRAY);
		IplImage *image = cvCreateImage(cvSize(512, 512), IPL_DEPTH_8U, 1); 
		cvResize(gray, image);

		IplImage* h_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);
		IplImage* d_image2 = cvCreateImage(cvGetSize(image), IPL_DEPTH_8U, 1);

		unsigned char *output = (unsigned char*)h_image2->imageData;
		unsigned char *input = (unsigned char*)image->imageData;
	

		//copiamos el input al device
		hipMemcpy(d_input, input, 512*512, hipMemcpyHostToDevice);
		//32*16 = 512 deber�amos soportar hasta 128x128,512x512,3072x3072,4096x4096
		dim3 dimBlock(32, 32);//x = 8
		dim3 dimGrid(16, 16);

		float milis;
		CUDA_TIME_START();
		sobelBlocks << <dimGrid, dimBlock >> > (d_input, d_output, cvGetSize(image).height);
		CUDA_TIME_GET(milis);

		CudaCheckError();
		hipMemcpy(output, d_output, 512*512, hipMemcpyDeviceToHost);
		CPUAscii((unsigned char*)h_image2->imageData, cvGetSize(image).height, 128, 32);
	}
	hipFree(d_output);
	hipFree(d_input);

}

int main()
{
	
	//createVideoAscii();
	createVideoAscii();
	/*printf("CPU_SOBEL: %f\n", serial());
	printf("Time milis: %f\n",cuda128()	);
	printf("Time milis: %f\n",cuda128_4()	);
	printf("Time milis: %f\n",cuda512()	);
	printf("Time milis: %f\n",cuda512_4()	);
	printf("Time milis: %f\n",cuda3072()	);
	printf("Time milis: %f\n",cuda3072_4());
	printf("Time milis: %f\n",cuda4096()	);
	printf("Time milis: %f\n",cuda4096_4());*/
	//int f;
	//std::cin >> f;
	return 0;
}

